#include "hip/hip_runtime.h"
/*
 * scheduler.cu
 *
 *  Created on: Jun 21, 2018
 *      Author: jsong
 */


/*
 * scheduler.cu
 *
 *  Created on: Jul 5, 2017
 *      Author: pfc
 */


#include <stdint.h>
#include "scheduler.h"
//#include "req_receiver.h"

//__shared__ uint16_t ia_rr_pointer[NUM_SWITCH_SIZE][NUM_RR_SEQ_SIZE];
//__shared__ uint16_t oa_rr_pointer[NUM_SWITCH_SIZE][NUM_RR_SEQ_SIZE];

uint16_t * ia_rr_pointer;
uint16_t * oa_rr_pointer;


uint32_t * device_voq_all;
uint32_t * host_voq_all;
uint32_t * VoQCount;

uint16_t * device_granted_input;
uint16_t * host_granted_input;
uint16_t * device_granted_output;
uint16_t * host_granted_output;
uint16_t * req_map;

__shared__ int sh_switch_size;
__shared__ int sh_block_size;
__shared__ int sh_scale_factor;
__shared__ int sh_num_bits_per_voq;
__shared__ int sh_num_req_per_uint32;
__shared__ int sh_num_rr_seq;



int _switch_size;
int _block_size;
int _scale_factor;
int _num_bits_per_voq;
int _num_req_per_uint32;
int _num_rr_seq;
int _num_iterations = 1;
int _num_cuda_blk = 4;
int _num_cuda_thread;



extern int eth_socket_init (int    argc, char **argv);



static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

void verify_grant()
{

	for (int i=0; i<_switch_size; i++)
		for (int j=1; j<_switch_size; j++)
		{
			short i1, i2, o1, o2;
			i1 = host_granted_input[i];
			i2 = host_granted_input[(i+j)%_switch_size];
			if (( i1== i2) & (i1 != -1 || i2 != -1))
			{
				printf("SW ERROR Duplicated granted_input[%d]:%d granted_input[%d]:%d \n", i, i1, (i+j)%_switch_size, i2);

			}

			o1 = host_granted_output[i];
			o2 = host_granted_output[(i+j)%_switch_size];
			if (( o1== o2) & (o1 != -1 || o2 != -1))
			{
				printf("SW ERROR Duplicated granted_output[%d]:%d granted_output[%d]:%d \n", i, o1, (i+j)%_switch_size, o2);
			}


		}
}




void copy_host_grant_reset_dev_grant()
{
	CUDA_CHECK_RETURN(hipMemcpy(host_granted_input, device_granted_input, sizeof(uint16_t)*_switch_size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(host_granted_output, device_granted_output, sizeof(uint16_t)*_switch_size, hipMemcpyDeviceToHost));
	hipMemset (device_granted_input, 0xff, sizeof(uint16_t)*_switch_size);
	hipMemset (device_granted_output, 0xff, sizeof(uint16_t)*_switch_size);
}

void reset_voq(int in_idx, int out_idx)
{
	int idx_h = out_idx/8;
	int idx_l = out_idx%8;
	uint32_t bit = 0xf0000000>>(4*idx_l);

	host_voq_all[in_idx*_block_size+idx_h] = host_voq_all[in_idx*_block_size+idx_h]&(~bit);
}

void update_voq()
{

	//CUDA_CHECK_RETURN(hipMemcpy(device_grant_map, host_grant_map, sizeof(uint16_t),hipMemcpyHostToDevice));

	for (int i=0; i<_switch_size; i++)
	{
		if (host_granted_output[i] == 0xffff)
			continue;

		if (VoQCount[i*_switch_size+host_granted_output[i]]==0)
			printf("SW ERROR input:%d output:%d count:%d !!!!!\n", i, host_granted_output[i], VoQCount[i*_switch_size+host_granted_output[i]]);
		VoQCount[i*_switch_size+host_granted_output[i]]--;

		if (VoQCount[i*_switch_size+host_granted_output[i]] == 0)
		{
			reset_voq(i, host_granted_output[i]);
		}
	}
}





void reset_req()
{
	hipMemset (req_map, 0xff, sizeof(uint16_t)*_switch_size*_switch_size);
}

__device__ void print_req_device (int idx, uint16_t * req_map )
{
	//printf ("Output %d \n", idx);

	for (int i=0; i<sh_switch_size; i++)
	{
		if (req_map[i*sh_switch_size+idx] == 0xffff)
			continue;
		printf ("Req(%d->%d):%d ",i, idx, req_map[i*sh_switch_size+idx]);
	}
	printf ("\n");
}

__global__ void print_req_global(uint16_t * req_map)
{
	int out_idx = blockIdx.x*blockDim.x+threadIdx.x;;
    print_req_device (out_idx, req_map);
}



__device__ void print_voq_device (int idx, uint32_t * voq )
{

	for (int i=0; i<sh_block_size; i++)
	{
		uint32_t mask = 0xf0000000;
		int out_idx_h = i>>3;
		for (int j=0; j<8; j++)
		{
			if (mask&voq[idx*sh_block_size+out_idx_h])
			{
				printf("(%d->%d):1 ", idx, out_idx_h*8+j);
			}
			else
			{
				printf("(%d->%d):0 ", idx, out_idx_h*8+j);

			}
			mask = mask>>sh_num_bits_per_voq; //NUM_BITS_PER_VOQ_INFO;
		}
	}
	printf(" \n");

}

__global__ void print_voq_global(uint32_t * voq)
{
	int in_idx = blockIdx.x*blockDim.x+threadIdx.x;;
    print_voq_device (in_idx, voq);
}



void print_req(uint16_t * req_map)
{
	print_req_global<<<_num_cuda_blk,_num_cuda_thread>>>(req_map);
}

__device__ void device_send_request(int in_idx,  uint32_t * voq_map, int ts, uint16_t * req_map, uint16_t* granted_input, uint16_t * rr_ptr)
{
	int rr_start = rr_ptr[in_idx*sh_num_rr_seq+ts%sh_num_rr_seq];

	int rr_start_h = rr_start>>3;
	int rr_start_l = rr_start%sh_num_req_per_uint32;

	uint32_t mask = 0xf0000000>>(rr_start_l*sh_num_bits_per_voq);
	if (voq_map[in_idx*sh_block_size+rr_start_h])
	{
		for (int j=rr_start_l; j<sh_num_req_per_uint32; j++)
		{
			//printf("input:%d mask:0x%08x sh_num_req_per_uint32:%d rr:%d rr_h:%d rr_l:%d \n", in_idx, mask, sh_num_req_per_uint32, rr_start, rr_start_h, rr_start_l);

			if ((mask&voq_map[in_idx*sh_block_size+rr_start_h])&&
				(granted_input[in_idx*sh_switch_size+rr_start_h*sh_num_req_per_uint32+j]!=0xffff))
			{
				req_map[in_idx*sh_switch_size+rr_start_h*sh_num_req_per_uint32+j] = 1;
				//printf("REQ#1 -- rr_ptr:%d, idx_l:%d send request from %d to %d \n", rr_start, rr_start_l, in_idx, rr_start_h*8+j);
				return;
			}
			mask = mask>>sh_num_bits_per_voq;
		}
	}

	for (int i=1; i<sh_block_size+1; i++)
	{
		int out_idx_h = ((rr_start_h+i)%sh_switch_size)>>3;
		if (voq_map[in_idx*sh_block_size+out_idx_h] == 0)
			continue;
		mask = 0xf0000000;
		for (int j=0; j<sh_num_req_per_uint32; j++)
		{
			//printf("input:%d mask:0x%08x  \n", in_idx, mask);
			if (mask&voq_map[in_idx*sh_block_size+out_idx_h])
			{
				req_map[in_idx*sh_switch_size+out_idx_h*sh_num_req_per_uint32+j] = 1;
				//printf("REQ#2 -- rr_ptr:%d, idx_l:%d send request from %d to %d \n", rr_start, rr_start_l, in_idx, out_idx_h*sh_num_req_per_uint32+j);
				return;
			}
			mask = mask>>sh_num_bits_per_voq;

		}
	}
	//printf("No request from %d, rr_start:%d \n", in_idx, rr_start);

}

__global__ void cuda_send_request(uint32_t * voq_map, int ts, uint16_t * granted_input, uint16_t * granted_output,uint16_t * req_map, uint16_t * rr_ptr)
{
	int input_idx = blockIdx.x*blockDim.x+threadIdx.x;
	//printf("Send Request for input:%04d \n", input_idx);

	if (granted_output[input_idx] != 0xffff)
	{
		printf("already granted for input:%d, ouput:%d \n", input_idx, granted_output[input_idx]);
		return;
	}
    device_send_request(input_idx, voq_map, ts,req_map, granted_input, rr_ptr);
}

__device__ void device_send_grant(int out_idx, int ts, uint16_t * req_map, uint16_t * granted_input, uint16_t * granted_output, uint16_t * rr_ptr)
{
	int rr_start = rr_ptr[out_idx*sh_num_rr_seq+ts%sh_num_rr_seq];


	for (int i=0; i<sh_switch_size; i++)
	{
		int in_idx = (rr_start+i)%sh_switch_size;
		if (req_map[in_idx*sh_switch_size+out_idx] == 0xffff)
			continue;

		granted_output [in_idx] = out_idx;
		granted_input [out_idx] = in_idx;

		return;
	}
}
__global__ void cuda_send_grant (int ts, uint16_t * req_map, uint16_t * granted_input, uint16_t * granted_output, uint16_t * rr_ptr)
{
	int output_idx = blockIdx.x*blockDim.x+threadIdx.x;
	device_send_grant(output_idx, ts, req_map, granted_input, granted_output, rr_ptr);
}

__global__ void cuda_cleanup_for_new_iter ()
{
	;
}

void print_voq(uint32_t * voq)
{
	for (int i=0; i<_switch_size; i++)
	{
		printf ("  VoQ for input %d: ", i);

		for (int j=0; j<_block_size; j++)
		{
			printf ("0x%08x ", voq[i*_block_size+j]);
		}
		printf ("\n");

	}
}




__device__ void init_rr_pointer_device (int idx, uint16_t * i_rr_ptr, uint16_t * o_rr_ptr )
{
	int off_set = idx*sh_num_rr_seq;
	for (int i=0; i<sh_num_rr_seq; i++)
	{
		i_rr_ptr[off_set+i] = (i+idx)%sh_switch_size;
		o_rr_ptr[off_set+i] = (sh_switch_size+i-idx)%sh_switch_size;
		//printf ("idx:%d i:%d, ,off_set:%d, i_rr:%d, o_rr:%d \n",idx, i, off_set, i_rr_ptr[idx*sh_switch_size+i], o_rr_ptr[idx*sh_switch_size+i]);
	}
	/*printf ("idx:%d off_set:%d sh_num_rr_seq:%d sh_switch_size:%d - i_rr: %d %d %d %d %d %d %d %d ..o_rr: %d %d %d %d %d %d %d %d ..\n",
														idx, off_set, sh_switch_size, sh_num_rr_seq,
														i_rr_ptr[off_set+0],
														i_rr_ptr[off_set+1],
														i_rr_ptr[off_set+2],
														i_rr_ptr[off_set+3],
														i_rr_ptr[off_set+4],
														i_rr_ptr[off_set+5],
														i_rr_ptr[off_set+6],
														i_rr_ptr[off_set+7],

														o_rr_ptr[off_set+0],
														o_rr_ptr[off_set+1],
														o_rr_ptr[off_set+2],
														o_rr_ptr[off_set+3],
														o_rr_ptr[off_set+4],
														o_rr_ptr[off_set+5],
														o_rr_ptr[off_set+6],
														o_rr_ptr[off_set+7]);*/
}

__global__ void init_rr_pointer(uint16_t * i_rr_ptr, uint16_t * o_rr_ptr )
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;;
	init_rr_pointer_device (idx, i_rr_ptr, o_rr_ptr );
}

__device__ void print_rr_pointer_device (int idx, uint16_t * i_rr_ptr)
{

	//printf ("idx:%d - %d %d %d %d %d %d ...\n", idx, i_rr_ptr->pointer[idx][0], i_rr_ptr->pointer[idx][1], i_rr_ptr->pointer[idx][2], i_rr_ptr->pointer[idx][3], i_rr_ptr->pointer[idx][4], i_rr_ptr->pointer[idx][5]);
}

__global__ void print_rr_pointer(uint16_t * i_rr_ptr)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;;
	print_rr_pointer_device (idx, i_rr_ptr);
}

void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

int pefrom_scheduling (int ts)
{
	CUDA_CHECK_RETURN(hipMemcpy(device_voq_all, host_voq_all, sizeof(uint32_t)*_switch_size*_block_size,hipMemcpyHostToDevice));

	for (int iter=0; iter<_num_iterations; iter++)
	{
    	reset_req();
    	//print_req(req_map);
    	cuda_send_request<<<_num_cuda_blk,_num_cuda_thread>>>(device_voq_all, ts, device_granted_input, device_granted_output, req_map, ia_rr_pointer);
    	//hipDeviceSynchronize();
		//print_req(req_map);
		cuda_send_grant <<<_num_cuda_blk,_num_cuda_thread>>>(ts, req_map, device_granted_input, device_granted_output, oa_rr_pointer);
		hipDeviceSynchronize();
		cuda_cleanup_for_new_iter <<<_num_cuda_blk,_num_cuda_thread>>>();

	}
	copy_host_grant_reset_dev_grant();

	return 0;
}

void copyMsgVoQToDevice (msgRequest_t * req)
{
	int in_idx = req->s_pfwi_id-1;
    CUDA_CHECK_RETURN(hipMemcpy((void *)&req->voq_info, (void *)&device_voq_all[in_idx*_block_size], sizeof(uint32_t)*_block_size, hipMemcpyHostToDevice));
	return;
}



void generate_packet(int load)
{
	int random_port;
	int toss;
	for (int i=0; i<_switch_size; i++)
	{
		toss = rand()%100;
		if (toss < load)
		{
			random_port = rand()%_switch_size;
			if (VoQCount[i*_switch_size+random_port] == 0)
			{
				int idx_h = random_port/8;
				int idx_l = random_port%8;
				uint32_t bit = 0x10000000>>(idx_l*4);
				if (host_voq_all[i*_block_size+idx_h]&bit)
				{
					printf("SW Error idx:%d idx_h:%d, idx_l:%d, count:%d, voq:0x%x \n",random_port, idx_h, idx_l, VoQCount[i*_switch_size+random_port], host_voq_all[i*_block_size+idx_h]);
				}
				host_voq_all[i*_block_size+idx_h] =  host_voq_all[i*_block_size+idx_h]|bit;
			}
			if (VoQCount[i*_switch_size+random_port] < NUM_VOQ_BUFFER_SIZE)
			{
				VoQCount[i*_switch_size+random_port] ++;
			}
		//printf("Gen Packet %d->%d \n",i, random_port);
		}
	}
}

__global__ void init_shared_value(int _switch_size, int _block_size, int _num_rr_seq, int _num_req_per_uint32, int _num_bits_per_voq)
{
	sh_switch_size = _switch_size;
	sh_block_size = _block_size;
	sh_num_rr_seq = _num_rr_seq;
	sh_num_req_per_uint32 = _num_req_per_uint32;
	sh_num_bits_per_voq = _num_bits_per_voq;
	//printf("Init shared value sh_switch_size:%d, sh_block_size:%d, sh_num_rr_seq:%d, sh_num_req_per_uint32:%d \n", sh_switch_size, sh_block_size, sh_num_rr_seq, sh_num_req_per_uint32);
}

__global__ void print_shared_value()
{
	printf("shared value sh_switch_size:%d, sh_block_size:%d, sh_num_rr_seq:%d \n", sh_switch_size, sh_block_size, sh_num_rr_seq);
}

void init_scheduler()
{
	CUDA_CHECK_RETURN(hipDeviceReset());
	_scale_factor = 2;
	_num_bits_per_voq = 4;
	_num_rr_seq = _switch_size*_scale_factor;
	_num_req_per_uint32 = 32/_num_bits_per_voq;
	_block_size = _switch_size/_num_req_per_uint32;
	printf("Init Values scale_factor:%d, num_bits_per_voq:%d, _num_rr_seq:%d, _num_req_per_uint32:%d, _block_size:%d \n",
			_scale_factor, _num_bits_per_voq, _num_rr_seq, _num_req_per_uint32, _block_size);

	init_shared_value<<<_num_cuda_blk,_num_cuda_thread>>>(_switch_size, _block_size, _num_rr_seq, _num_req_per_uint32, _num_bits_per_voq);
	CUDA_CHECK_RETURN(hipMalloc((void **) &device_voq_all, sizeof(uint32_t)*_switch_size*_block_size));
	CUDA_CHECK_RETURN(hipMemset ((void *) device_voq_all, 0, sizeof(uint32_t)*_switch_size*_block_size));

	if (host_voq_all)
		free(host_voq_all);
	host_voq_all = (uint32_t *) malloc (sizeof(uint32_t)*_switch_size*_block_size);
	memset(host_voq_all,0x0, sizeof(uint32_t)*_switch_size*_block_size);
	CUDA_CHECK_RETURN(hipMalloc ((void **) &device_granted_input, sizeof(uint16_t)*_switch_size));
	CUDA_CHECK_RETURN(hipMemset((void*) device_granted_input, 0xff, sizeof(uint16_t)*_switch_size));

	CUDA_CHECK_RETURN(hipMalloc ((void **) &device_granted_output, sizeof(uint16_t)*_switch_size));
	CUDA_CHECK_RETURN(hipMemset((void*) device_granted_output, 0xff, sizeof(uint16_t)*_switch_size));

	if (host_granted_input)
		free(host_granted_input);
	host_granted_input = (uint16_t *) malloc (sizeof(uint16_t)*_switch_size);
	memset(host_granted_input,0xff, sizeof(uint16_t)*_switch_size);


	if (host_granted_output)
		free(host_granted_output);
	host_granted_output = (uint16_t *) malloc (sizeof(uint16_t)*_switch_size);
	memset(host_granted_output,0xff, sizeof(uint16_t)*_switch_size);


	CUDA_CHECK_RETURN(hipMalloc ((void **) &req_map, sizeof(uint16_t)*_switch_size*_switch_size));

	CUDA_CHECK_RETURN(hipMalloc ((void **) &ia_rr_pointer, sizeof(uint16_t)*_switch_size*_num_rr_seq));
	CUDA_CHECK_RETURN(hipMalloc ((void **) &oa_rr_pointer, sizeof(uint16_t)*_switch_size*_num_rr_seq));
	init_rr_pointer<<<_num_cuda_blk,_num_cuda_thread>>>(ia_rr_pointer, oa_rr_pointer );

	if (VoQCount)
		free(VoQCount);
	VoQCount = (uint32_t *) malloc (sizeof(uint32_t)*_switch_size*_switch_size);
	memset(VoQCount,0x0, sizeof(uint32_t)*_switch_size*_switch_size);

}



int main(void)
{
	FILE * SimOutFile;
	FILE * SimOutFile2;
    int devCount = 0;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    if (devCount == 0)
	{
		std::cout<<"devCount : " << devCount << "  --> No GPU installed " <<std::endl;
		exit(1);
	}
    //hipSetDevice(0);

    // Iterate through devices

    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA  #%d \n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }

    hipSetDevice(1);
	/*if (argc < 4)
	{
		std::cout<<" argc " << argc<< "-- exit " <<std::endl;
		std::cout<<"More Arguments required - duration load iterations" <<argc<<std::endl;
		exit(1);
	}


	int _duration = atoi(argv[1]);
	int _load = atoi(argv[2]);
	int _iteration = atoi(argv[3]);
	*/
    int _duration = 1000;
    int _load = 90;
    int _ts_th = -1;
	int _q_th = -1;
	int _aware = 1;

	uint64_t sum_delay, measure_count, max_delay, min_delay, measure_start;

	char fname[60];
	char fname2[60];

	sprintf(fname,"sim_result_scheduling_time_block_%d.txt", _num_cuda_blk);
	SimOutFile = fopen(fname,"w");
	printf("open fname %s\n", fname);

	for (_load = 50; _load<51; _load ++ )
	for (_switch_size = 8; _switch_size<2000; _switch_size ++ )

	{
		/*
			receiver_req function takes ethernet stream
		*/
		//receive_req(_switch_size);
		

		if (_load%10 != 0 || _switch_size%_num_cuda_blk != 0 || _switch_size%8 != 0)
		{
			//printf ("continue switch_size %d, duration %d,  load %d, iteration %d,  _ts_th %d, _q_th %d, _aware %d \n",
			//					_switch_size, _duration, _load, _num_iterations,  _ts_th, _q_th, _aware);

			continue;
		}

		sprintf(fname2,"sim_result_load_%d_switch_%d_block_%d.txt", _load, _switch_size, _num_cuda_blk);
		SimOutFile2 = fopen(fname2,"w");
		printf("open fname2 %s\n", fname2);

		sum_delay=0;
		measure_count = 0;
		measure_start = 0;
		max_delay = 0;
		min_delay = 1000000;
		_num_cuda_thread = _switch_size/_num_cuda_blk;
		init_scheduler();

		for (int i=0; i<_duration; i++)
		{
			// process request
			if (i==0)
				measure_start = 0;

			int _time_slot = i;
			//printf("timeslot %d \n", _time_slot);
			//print_rr_pointer <<<1,_switch_size>>>();


			// reset_req(req_map);



			generate_packet(_load);

			//print_voq(host_voq_all);
			//print_rr_pointer<<<1,_switch_size>>>(ia_rr_pointer);


			//printf("After hipMalloc host_voq ---\n");
			//print_voq(host_voq_all );
			//printf("After hipMalloc devicet_voq ---\n");



			//print_voq_global <<<1,_switch_size>>>(device_voq_all);
			//hipDeviceSynchronize();

			clock_t t1 = clock();

			pefrom_scheduling ( _time_slot);

			clock_t t2 = clock();

			clock_t diff = t2-t1;
			int schedule_time_usec = diff*1000000/CLOCKS_PER_SEC;
			//printf("Delta t2-t1: %d \n \n", schedule_time_usec);

			if (i==100)
				measure_start = 1;
			if (measure_start)
			{
				if (diff > max_delay)
					max_delay = diff;
				if (diff < min_delay)
					min_delay = diff;

				sum_delay+= diff;
				measure_count++;
				fprintf(SimOutFile2, "delay %d\n", diff);

			}

			update_voq();  // THis operation is excluded for processing time because this happens in linecards
			//hipDeviceSynchronize();

		}
		fprintf(SimOutFile, "switch_size %d load %d iteration %d max_delay %lu min_delay %lu sum_delay %lu measure_count %lu\n",
				_switch_size,_load, _num_iterations, max_delay, min_delay, sum_delay, measure_count);
		printf("switch_size %d load %d iteration %d max_delay %lu min_delay %lu sum_delay %lu measure_count %lu\n",
				_switch_size,_load, _num_iterations, max_delay, min_delay, sum_delay, measure_count);
		fclose(SimOutFile2);

	}
	fclose(SimOutFile);
	printf("close %s %d\n", fname, SimOutFile);
	return 0;
}







