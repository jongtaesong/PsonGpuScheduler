#include "hip/hip_runtime.h"
/*
 * scheduler.cu
 *
 *  Created on: Jun 21, 2018
 *      Author: jsong
 */





#include <stdint.h>
#include "scheduler.h"
//#include "req_receiver.h"

//__shared__ uint16_t ia_rr_pointer[NUM_SWITCH_SIZE][NUM_RR_SEQ_SIZE];
//__shared__ uint16_t oa_rr_pointer[NUM_SWITCH_SIZE][NUM_RR_SEQ_SIZE];

uint16_t * ia_rr_pointer;
uint16_t * oa_rr_pointer;


uint8_t * device_voq_all;
uint8_t * host_voq_all;
uint32_t * VoQCount;

uint32_t * device_granted_input;
uint32_t * host_granted_input;
uint32_t * device_granted_output;
uint32_t * host_granted_output;
uint16_t * req_map;

__shared__ int sh_switch_size;
__shared__ int sh_scale_factor;
__shared__ int sh_num_rr_seq;



int _switch_size;
int _scale_factor;
int _num_rr_seq;
int _num_iterations = 1;
int _num_cuda_blk = 4;
int _num_cuda_thread;



extern int eth_socket_init (int    argc, char **argv);
extern void receive_req(int sw_size);



static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

void verify_grant()
{

	for (int i=0; i<_switch_size; i++)
		for (int j=1; j<_switch_size; j++)
		{
			short i1, i2, o1, o2;
			i1 = host_granted_input[i];
			i2 = host_granted_input[(i+j)%_switch_size];
			if (( i1== i2) & (i1 != -1 || i2 != -1))
			{
				printf("SW ERROR Duplicated granted_input[%d]:%d granted_input[%d]:%d \n", i, i1, (i+j)%_switch_size, i2);
			}

			o1 = host_granted_output[i];
			o2 = host_granted_output[(i+j)%_switch_size];
			if (( o1== o2) & (o1 != -1 || o2 != -1))
			{
				printf("SW ERROR Duplicated granted_output[%d]:%d granted_output[%d]:%d \n", i, o1, (i+j)%_switch_size, o2);
			}
		}
}


void copy_host_grant_reset_dev_grant()
{
	CUDA_CHECK_RETURN(hipMemcpy(host_granted_input, device_granted_input, sizeof(uint32_t)*_switch_size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(host_granted_output, device_granted_output, sizeof(uint32_t)*_switch_size, hipMemcpyDeviceToHost));
	hipMemset (device_granted_input, 0x0, sizeof(uint16_t)*_switch_size);
	hipMemset (device_granted_output, 0x0, sizeof(uint16_t)*_switch_size);
}

void reset_voq(int in_idx, int out_idx)
{
	host_voq_all[in_idx*_switch_size+out_idx] = 0;
}

void update_voq()
{

	for (int i=0; i<_switch_size; i++)
	{
		if (host_granted_output[i] == 0)
			continue;

		if (VoQCount[i*_switch_size+host_granted_output[i]]==0)
			printf("SW ERROR input:%d output:%d count:%d !!!!!\n", i, host_granted_output[i], VoQCount[i*_switch_size+host_granted_output[i]]);
		VoQCount[i*_switch_size+host_granted_output[i]]--;

		if (VoQCount[i*_switch_size+host_granted_output[i]] == 0)
		{
			reset_voq(i, host_granted_output[i]);
		}
	}
}

void reset_req()
{
	hipMemset (req_map, 0, sizeof(uint16_t)*_switch_size*_switch_size);
}

__device__ void print_req_device (int idx, uint16_t * req_map )
{
	//printf ("Output %d \n", idx);

	for (int i=0; i<sh_switch_size; i++)
	{
		if (req_map[i*sh_switch_size+idx] == 0)
			continue;
		printf ("Req(%d->%d):%d ",i, idx, req_map[i*sh_switch_size+idx]);
	}
	printf ("\n");
}

__global__ void print_req_global(uint16_t * req_map)
{
	int out_idx = blockIdx.x*blockDim.x+threadIdx.x;;
    print_req_device (out_idx, req_map);
}

__device__ void print_voq_device (int idx, uint8_t * voq )
{
	for (int i=0; i<sh_switch_size; i++)
	{
		printf("(%d->%d):%d ", idx, i, voq[i]);
	}
	printf(" \n");
}

__global__ void print_voq_global(uint8_t * voq)
{
	int in_idx = blockIdx.x*blockDim.x+threadIdx.x;;
    print_voq_device (in_idx, voq);
}

void print_req(uint16_t * req_map)
{
	print_req_global<<<_num_cuda_blk,_num_cuda_thread>>>(req_map);
}

__device__ void device_send_request(int in_idx,  uint8_t * voq_map, int ts, uint16_t * req_map, uint32_t* granted_input, uint16_t * rr_ptr)
{
	int rr_start = rr_ptr[in_idx*sh_num_rr_seq+ts%sh_num_rr_seq];

	for (int i=0; i<sh_switch_size; i++)
	{
		int out_idx = (rr_start+i)%sh_switch_size;
		if (voq_map[in_idx*sh_switch_size+out_idx] == 0)
			continue;

		req_map[in_idx*sh_switch_size+out_idx] = 1;
		return;
	}
}

__global__ void cuda_send_request(uint8_t * voq_map, int ts, uint32_t * granted_input, uint32_t * granted_output,uint16_t * req_map, uint16_t * rr_ptr)
{
	int input_idx = blockIdx.x*blockDim.x+threadIdx.x;
	//printf("Send Request for input:%04d \n", input_idx);

	if (granted_output[input_idx] != 0)
	{
		printf("already granted for input:%d, ouput:%d \n", input_idx, granted_output[input_idx]);
		return;
	}
    device_send_request(input_idx, voq_map, ts,req_map, granted_input, rr_ptr);
}

__device__ void device_send_grant(int out_idx, int ts, uint16_t * req_map, uint32_t * granted_input, uint32_t * granted_output, uint16_t * rr_ptr)
{
	int rr_start = rr_ptr[out_idx*sh_num_rr_seq+ts%sh_num_rr_seq];

	for (int i=0; i<sh_switch_size; i++)
	{
		int in_idx = (rr_start+i)%sh_switch_size;
		if (req_map[in_idx*sh_switch_size+out_idx] == 0)
			continue;

		granted_output [in_idx] = out_idx;
		granted_input [out_idx] = in_idx;

		return;
	}
}
__global__ void cuda_send_grant (int ts, uint16_t * req_map, uint32_t * granted_input, uint32_t * granted_output, uint16_t * rr_ptr)
{
	int output_idx = blockIdx.x*blockDim.x+threadIdx.x;
	device_send_grant(output_idx, ts, req_map, granted_input, granted_output, rr_ptr);
}

__global__ void cuda_cleanup_for_new_iter ()
{
	;
}

void print_voq(uint8_t * voq)
{
	for (int i=0; i<_switch_size; i++)
	{
		printf ("  VoQ for input %d: ", i);

		for (int j=0; j<_switch_size; j++)
		{
			printf ("0x%02x ", voq[i*_switch_size+j]);
		}
		printf ("\n");
	}
}

__device__ void init_rr_pointer_device (int idx, uint16_t * i_rr_ptr, uint16_t * o_rr_ptr )
{
	int off_set = idx*sh_num_rr_seq;
	for (int i=0; i<sh_num_rr_seq; i++)
	{
		i_rr_ptr[off_set+i] = (i+idx)%sh_switch_size;
		o_rr_ptr[off_set+i] = (sh_switch_size+i-idx)%sh_switch_size;
		//printf ("idx:%d i:%d, ,off_set:%d, i_rr:%d, o_rr:%d \n",idx, i, off_set, i_rr_ptr[idx*sh_switch_size+i], o_rr_ptr[idx*sh_switch_size+i]);
	}
}

__global__ void init_rr_pointer(uint16_t * i_rr_ptr, uint16_t * o_rr_ptr )
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;;
	init_rr_pointer_device (idx, i_rr_ptr, o_rr_ptr );
}

__device__ void print_rr_pointer_device (int idx, uint16_t * i_rr_ptr)
{
	//printf ("idx:%d - %d %d %d %d %d %d ...\n", idx, i_rr_ptr->pointer[idx][0], i_rr_ptr->pointer[idx][1], i_rr_ptr->pointer[idx][2], i_rr_ptr->pointer[idx][3], i_rr_ptr->pointer[idx][4], i_rr_ptr->pointer[idx][5]);
}

__global__ void print_rr_pointer(uint16_t * i_rr_ptr)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;;
	print_rr_pointer_device (idx, i_rr_ptr);
}

void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

int pefrom_scheduling (int ts)
{
	CUDA_CHECK_RETURN(hipMemcpy(device_voq_all, host_voq_all, sizeof(uint8_t)*_switch_size*_switch_size,hipMemcpyHostToDevice));

	for (int iter=0; iter<_num_iterations; iter++)
	{
    	reset_req();
    	//print_req(req_map);
    	cuda_send_request<<<_num_cuda_blk,_num_cuda_thread>>>(device_voq_all, ts, device_granted_input, device_granted_output, req_map, ia_rr_pointer);
    	//hipDeviceSynchronize();
		//print_req(req_map);
		cuda_send_grant <<<_num_cuda_blk,_num_cuda_thread>>>(ts, req_map, device_granted_input, device_granted_output, oa_rr_pointer);
		hipDeviceSynchronize();
		cuda_cleanup_for_new_iter <<<_num_cuda_blk,_num_cuda_thread>>>();

	}
	copy_host_grant_reset_dev_grant();

	return 0;
}

void copyMsgVoQToDevice (msgRequest_t * req)
{
	int in_idx = req->s_pfwi_id-1;
    CUDA_CHECK_RETURN(hipMemcpy((void *)&req->voq_info, (void *)&device_voq_all[in_idx*_switch_size], sizeof(uint8_t)*_switch_size, hipMemcpyHostToDevice));
	return;
}



void generate_packet(int load)
{
	int random_port;
	int toss;
	for (int i=0; i<_switch_size; i++)
	{
		toss = rand()%100;
		if (toss < load)
		{
			random_port = rand()%_switch_size;
			if (VoQCount[i*_switch_size+random_port] == 0)
			{
				int idx = random_port;
				if (host_voq_all[i*_switch_size+idx])
				{
					printf("SW Error idx:%d, count:%d, voq:0x%x \n",random_port, VoQCount[i*_switch_size+random_port], host_voq_all[i*_switch_size+idx]);
				}
				host_voq_all[i*_switch_size+idx] =  1;
			}
			if (VoQCount[i*_switch_size+random_port] < NUM_VOQ_BUFFER_SIZE)
			{
				VoQCount[i*_switch_size+random_port] ++;
			}
		//printf("Gen Packet %d->%d \n",i, random_port);
		}
	}
}

__global__ void init_shared_value(int _switch_size, int _num_rr_seq)
{
	sh_switch_size = _switch_size;
	sh_num_rr_seq = _num_rr_seq;
	//printf("Init shared value sh_switch_size:%d, sh_block_size:%d, sh_num_rr_seq:%d, sh_num_req_per_uint32:%d \n", sh_switch_size, sh_block_size, sh_num_rr_seq, sh_num_req_per_uint32);
}

__global__ void print_shared_value()
{
	printf("shared value sh_switch_size:%d, sh_num_rr_seq:%d \n", sh_switch_size, sh_num_rr_seq);
}

void init_scheduler()
{
	CUDA_CHECK_RETURN(hipDeviceReset());
	_scale_factor = 2;
	_num_rr_seq = _switch_size*_scale_factor;
	printf("Init Values scale_factor:%d, num_rr_seq:%d \n", _scale_factor, _num_rr_seq );

	init_shared_value<<<_num_cuda_blk,_num_cuda_thread>>>(_switch_size, _num_rr_seq);
	CUDA_CHECK_RETURN(hipMalloc((void **) &device_voq_all, sizeof(uint8_t)*_switch_size*_switch_size));
	CUDA_CHECK_RETURN(hipMemset ((void *) device_voq_all, 0, sizeof(uint8_t)*_switch_size*_switch_size));

	if (host_voq_all)
		free(host_voq_all);
	host_voq_all = (uint8_t *) malloc (sizeof(uint8_t)*_switch_size*_switch_size);
	memset(host_voq_all,0x0, sizeof(uint8_t)*_switch_size*_switch_size);
	CUDA_CHECK_RETURN(hipMalloc ((void **) &device_granted_input, sizeof(uint16_t)*_switch_size));
	CUDA_CHECK_RETURN(hipMemset((void*) device_granted_input, 0, sizeof(uint16_t)*_switch_size));

	CUDA_CHECK_RETURN(hipMalloc ((void **) &device_granted_output, sizeof(uint16_t)*_switch_size));
	CUDA_CHECK_RETURN(hipMemset((void*) device_granted_output, 0, sizeof(uint16_t)*_switch_size));

	if (host_granted_input)
		free(host_granted_input);
	host_granted_input = (uint32_t *) malloc (sizeof(uint32_t)*_switch_size);
	memset(host_granted_input,0, sizeof(uint32_t)*_switch_size);

	if (host_granted_output)
		free(host_granted_output);
	host_granted_output = (uint32_t *) malloc (sizeof(uint32_t)*_switch_size);
	memset(host_granted_output,0, sizeof(uint32_t)*_switch_size);


	CUDA_CHECK_RETURN(hipMalloc ((void **) &req_map, sizeof(uint16_t)*_switch_size*_switch_size));

	CUDA_CHECK_RETURN(hipMalloc ((void **) &ia_rr_pointer, sizeof(uint16_t)*_switch_size*_num_rr_seq));
	CUDA_CHECK_RETURN(hipMalloc ((void **) &oa_rr_pointer, sizeof(uint16_t)*_switch_size*_num_rr_seq));
	init_rr_pointer<<<_num_cuda_blk,_num_cuda_thread>>>(ia_rr_pointer, oa_rr_pointer );

	if (VoQCount)
		free(VoQCount);
	VoQCount = (uint32_t *) malloc (sizeof(uint32_t)*_switch_size*_switch_size);
	memset(VoQCount,0x0, sizeof(uint32_t)*_switch_size*_switch_size);

}

void * start_receive_req_thread(void * arg)
{
	int * sw_size = (int *) (arg);
	receive_req(*sw_size);
	return 0;

}

int main(int argc, char * argv[])
{


    int devCount = 0;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);

    if (devCount == 0)
	{
		std::cout<<"devCount : " << devCount << "  --> No GPU installed " <<std::endl;
		exit(1);
	}
    //hipSetDevice(0);

    // Iterate through devices

    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA  #%d \n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }

    hipSetDevice(1);

	_num_cuda_thread = _switch_size/_num_cuda_blk;

#ifdef SIM
	FILE * SimOutFile;
	FILE * SimOutFile2;

    int _duration = 1000;
    int _load = 90;
    int _ts_th = -1;
	int _q_th = -1;
	int _aware = 1;

	uint64_t sum_delay, measure_count, max_delay, min_delay, measure_start;

	char fname[60];
	char fname2[60];

	sprintf(fname,"sim_result_scheduling_time_block_%d.txt", _num_cuda_blk);
	SimOutFile = fopen(fname,"w");
	printf("open fname %s\n", fname);

	for (_load = 50; _load<51; _load ++ )
	for (_switch_size = 8; _switch_size<2000; _switch_size ++ )

	{

		if (_load%10 != 0 || _switch_size%_num_cuda_blk != 0 || _switch_size%8 != 0)
		{
			continue;
		}

		sprintf(fname2,"sim_result_load_%d_switch_%d_block_%d.txt", _load, _switch_size, _num_cuda_blk);
		SimOutFile2 = fopen(fname2,"w");
		printf("open fname2 %s\n", fname2);

		sum_delay=0;
		measure_count = 0;
		measure_start = 0;
		max_delay = 0;
		min_delay = 1000000;
		_num_cuda_thread = _switch_size/_num_cuda_blk;
		init_scheduler();

		for (int i=0; i<_duration; i++)
		{
			// process request
			if (i==0)
				measure_start = 0;

			int _time_slot = i;

			generate_packet(_load);

			clock_t t1 = clock();

			pefrom_scheduling ( _time_slot);

			clock_t t2 = clock();

			clock_t diff = t2-t1;
			int schedule_time_usec = diff*1000000/CLOCKS_PER_SEC;
			//printf("Delta t2-t1: %d \n \n", schedule_time_usec);

			if (i==100)
				measure_start = 1;
			if (measure_start)
			{
				if (diff > max_delay)
					max_delay = diff;
				if (diff < min_delay)
					min_delay = diff;

				sum_delay+= diff;
				measure_count++;
				fprintf(SimOutFile2, "delay %d\n", diff);

			}

			update_voq();  // This operation is excluded for processing time because this happens in linecards
			//hipDeviceSynchronize();

		}
		fprintf(SimOutFile, "switch_size %d load %d iteration %d max_delay %lu min_delay %lu sum_delay %lu measure_count %lu\n",
				_switch_size,_load, _num_iterations, max_delay, min_delay, sum_delay, measure_count);
		printf("switch_size %d load %d iteration %d max_delay %lu min_delay %lu sum_delay %lu measure_count %lu\n",
				_switch_size,_load, _num_iterations, max_delay, min_delay, sum_delay, measure_count);
		fclose(SimOutFile2);

	}
	fclose(SimOutFile);
	printf("close %s %d\n", fname, SimOutFile);
	return 0;
#else
	if (argc < 3)
	{
		std::cout<<" argc " << argc<< "-- exit " <<std::endl;
		std::cout<<"More Arguments required - switch_size, iteration" <<argc<<std::endl;
		exit(1);
	}


	_switch_size = atoi(argv[1]);
	_num_iterations = atoi(argv[2]);


	init_scheduler();

	pthread_t       SynchProc_threadID, ReqProc_threadID;


	// create ReqProc_threadID
	if (pthread_create(&ReqProc_threadID, NULL, start_receive_req_thread, & _switch_size) != 0)
	{
		printf("ReqProc Failed [%s, %d, %s]\n", __FILE__, __LINE__, __FUNCTION__);
	} else
	{
		printf("ReqProc is started [%s, %d, %s]\n", __FILE__, __LINE__, __FUNCTION__);
	}

/*
	// SynchProc_thread
	if (pthread_create(&SynchProc_threadID, NULL, start_receive_synch_thread, & _switch_size) != 0)
	{
		printf("SynchProc Failed [%s, %d, %s]\n", __FILE__, __LINE__, __FUNCTION__);
	} else
	{
		printf("SynchProc is started [%s, %d, %s]\n", __FILE__, __LINE__, __FUNCTION__);
	}
*/


	return 0;

#endif

}







